#include "hip/hip_runtime.h"
// Copyright 2019 Emanuele Palazzolo (emanuele.palazzolo@uni-bonn.de), Cyrill Stachniss, University of Bonn
#include "tsdfvh/tsdf_volume.h"
#include <cfloat>
#include <cmath>
#include "marching_cubes/mesh_extractor.h"

#define THREADS_PER_BLOCK2 64

namespace refusion 
{

    namespace tsdfvh 
    {

        void eventCreate(Event *event) 
        {
            hipEventCreateWithFlags(&event->cpy_htd, eventFlags());
            hipEventCreateWithFlags(&event->compute, eventFlags());
            hipEventCreateWithFlags(&event->cpy_dth, eventFlags());
        }

        void eventDestroy(Event *event) 
        {
            hipEventDestroy(event->cpy_htd);
            hipEventDestroy(event->compute);
            hipEventDestroy(event->cpy_dth);
        }

        void eventSynchronize(Event *event) 
        {
            hipEventSynchronize(event->cpy_dth);
        }

        TsdfVolume::~TsdfVolume() 
        {
            hipDeviceSynchronize();

            gpuErrchk(hipStreamDestroy(int_stream_))
            gpuErrchk(hipStreamDestroy(mesh_stream_))

            for (auto x: render_streams_) 
                gpuErrchk(hipStreamDestroy(x))
            for (auto x: render_events_) 
            { 
                delete x; 
            }

            delete int_event_;

            gpuErrchk(hipFree(d_bgr_in_))
            gpuErrchk(hipFree(d_depth_in_))

            gpuErrchk(hipHostFree(h_bgr_in_))
            gpuErrchk(hipHostFree(h_depth_in_))

            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                gpuErrchk(hipFree(d_bgr_render_[i]))
                gpuErrchk(hipFree(d_instance_bgr_render_[i]))
                gpuErrchk(hipFree(d_depth_render_[i]))


                gpuErrchk(hipHostFree(std::get<0>(h_bgr_render_)[i]))
                gpuErrchk(hipHostFree(std::get<1>(h_bgr_render_)[i]))
                
                gpuErrchk(hipHostFree(std::get<0>(h_instance_bgr_render_)[i]))
                gpuErrchk(hipHostFree(std::get<1>(h_instance_bgr_render_)[i]))
                
                gpuErrchk(hipHostFree(std::get<0>(h_depth_render_)[i]))
                gpuErrchk(hipHostFree(std::get<1>(h_depth_render_)[i]))
            }
        }

        int TsdfVolume::GetInstanceNum()
        {
            return InstanceTable.size();
        }

        void TsdfVolume::UpdateInstanceTable(int InstanceIndex,int ClassIndex)
        {
            InstanceTable[InstanceIndex-1].ClassCount[ClassIndex] += 1;
        }

        void TsdfVolume::PrintInstanceTable()
        {
            int InstanceNum = GetInstanceNum();
            std::cout<<"InstanceTable:"<<InstanceNum<<std::endl;
            for(int i = 0;i<InstanceNum;i++)
            {
                std::cout<<"\t"<<i<<": "<<(unsigned int )InstanceTable[i].ClassIndex<<"  color:"<<int(InstanceTable[i].color[0])<<","<<int(InstanceTable[i].color[1])<<","<<int(InstanceTable[i].color[2])<<std::endl;

            }
        }

        void TsdfVolume::generateColorList()
        {
            // int a = 1, b = 255;
            // srand((int)time(NULL));
            unsigned char* InstanceColorList_cpu = (unsigned char*)malloc(GetInstanceNum() * 3 * sizeof(unsigned char));
            for (int i = 0; i < GetInstanceNum(); ++i)
            {
                InstanceColorList_cpu[i*3+0] = InstanceTable[i].color[0];
                InstanceColorList_cpu[i*3+1] = InstanceTable[i].color[1];
                InstanceColorList_cpu[i*3+2] = InstanceTable[i].color[2];
            }
            gpuErrchk(hipFree(InstanceColorList_gpu))
            hipMalloc((void **)&InstanceColorList_gpu, GetInstanceNum() * 3 * sizeof(unsigned char));
            hipMemcpy(InstanceColorList_gpu, InstanceColorList_cpu, GetInstanceNum() * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
            free(InstanceColorList_cpu);

        }

        void TsdfVolume::GetInstanceColor(unsigned int InstanceIndex,cv::Scalar& color)
        {
            color = cv::Scalar(InstanceTable[InstanceIndex].color[0],InstanceTable[InstanceIndex].color[1],InstanceTable[InstanceIndex].color[2]);
        }

        void TsdfVolume::InstanceTableAdd(unsigned char InstanceClass)
        {

            int a = 1, b = 255;
            // srand((int)time(NULL));

            // unsigned char* InstanceColor_cpu = (unsigned char*)malloc(sizeof(unsigned char)*3);
            InstanceBlock block;
            block.ClassIndex = InstanceClass;
            block.color = (unsigned char*)malloc(sizeof(unsigned char)*3);
            for (int i = 0; i < 3; ++i)
            {
                block.color[i] = a + rand() % (b - a + 1);
            }
            block.ClassCount = (short*)malloc(sizeof(short)*81);
            memset(block.ClassCount,0,sizeof(short)*81);
            block.ClassCount[InstanceClass]+=1;
            InstanceTable.push_back(block);
            generateColorList();
        }

        void TsdfVolume::Init(const TsdfVolumeOptions &options) 
        {
            options_ = options;
            HashTable::Init(options_.num_buckets, options_.bucket_size, options_.num_blocks, options_.block_size);

            should_call_next_ = "IntegrateScanAsync";

            // TODO(perf): Using low priority for now but maybe higher is better
            int least_prio, greatest_prio;
            gpuErrchk(hipDeviceGetStreamPriorityRange(&least_prio, &greatest_prio));

            gpuErrchk(hipStreamCreateWithPriority(&int_stream_, hipStreamNonBlocking, least_prio))
            gpuErrchk(hipStreamCreateWithPriority(&mesh_stream_, hipStreamNonBlocking, least_prio))
            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                hipStream_t render_stream;
                gpuErrchk(hipStreamCreateWithPriority(&render_stream, hipStreamNonBlocking, least_prio))
                render_streams_.push_back(render_stream);
                render_events_.push_back(nullptr);
            }

            gpuErrchk(hipMalloc(&d_bgr_in_, sizeof(uchar3) * num_pixels()))
            gpuErrchk(hipMalloc(&d_depth_in_, sizeof(float) * num_pixels()))

            gpuErrchk(hipHostMalloc(&h_bgr_in_, sizeof(uchar3) * num_pixels()))
            gpuErrchk(hipHostMalloc(&h_depth_in_, sizeof(float) * num_pixels()))

            uchar3 *h_bgr, *d_bgr;
            uchar3 *h_instance_bgr, *d_instance_bgr;
            float *h_depth, *d_depth;
            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                gpuErrchk(hipMalloc(&d_bgr, sizeof(uchar3) * num_pixels()))
                d_bgr_render_.push_back(d_bgr);

                gpuErrchk(hipMalloc(&d_instance_bgr, sizeof(uchar3) * num_pixels()))
                d_instance_bgr_render_.push_back(d_instance_bgr);

                gpuErrchk(hipMalloc(&d_depth, sizeof(float) * num_pixels()))
                d_depth_render_.push_back(d_depth);

                gpuErrchk(hipHostMalloc(&h_bgr, sizeof(uchar3) * num_pixels()))
                std::get<0>(h_bgr_render_).push_back(h_bgr);
                gpuErrchk(hipHostMalloc(&h_bgr, sizeof(uchar3) * num_pixels()))
                std::get<1>(h_bgr_render_).push_back(h_bgr);

                gpuErrchk(hipHostMalloc(&h_instance_bgr, sizeof(uchar3) * num_pixels()))
                std::get<0>(h_instance_bgr_render_).push_back(h_instance_bgr);
                gpuErrchk(hipHostMalloc(&h_instance_bgr, sizeof(uchar3) * num_pixels()))
                std::get<1>(h_instance_bgr_render_).push_back(h_instance_bgr);

                gpuErrchk(hipHostMalloc(&h_depth, sizeof(float) * num_pixels()))
                std::get<0>(h_depth_render_).push_back(h_depth);
                gpuErrchk(hipHostMalloc(&h_depth, sizeof(float) * num_pixels()))
                std::get<1>(h_depth_render_).push_back(h_depth);
            }
        }

        __host__ __device__ float3 TsdfVolume::GlobalVoxelToWorld(int3 position) 
        {
            return make_float3(position.x * options_.voxel_size,
                               position.y * options_.voxel_size,
                               position.z * options_.voxel_size);
        }

        __host__ __device__ int3 TsdfVolume::WorldToGlobalVoxel(float3 position) 
        {
            return make_int3(position.x / options_.voxel_size + signf(position.x) * 0.5f,
                             position.y / options_.voxel_size + signf(position.y) * 0.5f,
                             position.z / options_.voxel_size + signf(position.z) * 0.5f);
        }

        __host__ __device__ int3 TsdfVolume::WorldToBlock(float3 position) 
        {
            int3 voxel_position = WorldToGlobalVoxel(position);
            int3 block_position;
            if (voxel_position.x < 0)
                block_position.x = (voxel_position.x - block_size_ + 1) / block_size_;
            else
                block_position.x = voxel_position.x / block_size_;

            if (voxel_position.y < 0)
                block_position.y = (voxel_position.y - block_size_ + 1) / block_size_;
            else
                block_position.y = voxel_position.y / block_size_;

            if (voxel_position.z < 0)
                block_position.z = (voxel_position.z - block_size_ + 1) / block_size_;
            else
                block_position.z = voxel_position.z / block_size_;

            return block_position;
        }

        __host__ __device__ int3 TsdfVolume::WorldToLocalVoxel(float3 position) 
        {
            int3 position_global = WorldToGlobalVoxel(position);
            int3 position_local = make_int3(position_global.x % block_size_,
                                            position_global.y % block_size_,
                                            position_global.z % block_size_);
            if (position_local.x < 0) 
                position_local.x += block_size_;
            if (position_local.y < 0) 
                position_local.y += block_size_;
            if (position_local.z < 0) 
                position_local.z += block_size_;
            return position_local;
        }

        __host__ __device__ Voxel TsdfVolume::GetVoxel(float3 position) 
        {
            int3 block_position = WorldToBlock(position);
            int3 local_voxel = WorldToLocalVoxel(position);
            HashEntry entry = HashTable::FindHashEntry(block_position);
            if (entry.pointer == kFreeEntry) 
            {
                Voxel voxel;
                voxel.sdf = 0;
                voxel.color = make_uchar3(0, 0, 0);
                voxel.weight = 0;
                return voxel;
            }
            return HashTable::voxel_blocks_[entry.pointer].at(local_voxel);
        }

        __host__ __device__ Voxel TsdfVolume::GetInterpolatedVoxel(float3 position) 
        {
            Voxel v0 = GetVoxel(position);
            if (v0.weight == 0) 
                return v0;
            float voxel_size = options_.voxel_size;
            const float3 pos_dual = position -make_float3(voxel_size / 2.0f, voxel_size / 2.0f, voxel_size / 2.0f);
            float3 voxel_position = position / voxel_size;
            float3 weight = make_float3( voxel_position.x - floor(voxel_position.x),
                                                                        voxel_position.y - floor(voxel_position.y),
                                                                        voxel_position.z - floor(voxel_position.z));

            float distance = 0.0f;
            float3 color_float = make_float3(0.0f, 0.0f, 0.0f);
            float3 vColor;

            Voxel v = GetVoxel(pos_dual + make_float3(0.0f, 0.0f, 0.0f));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance +=(1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * v0.sdf;
                color_float = color_float +(1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * v.sdf;
                color_float = color_float + (1.0f - weight.x) * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(voxel_size, 0.0f, 0.0f));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += weight.x * (1.0f - weight.y) * (1.0f - weight.z) * v0.sdf;
                color_float = color_float + weight.x * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += weight.x * (1.0f - weight.y) * (1.0f - weight.z) * v.sdf;
                color_float = color_float + weight.x * (1.0f - weight.y) * (1.0f - weight.z) * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(0.0f, voxel_size, 0.0f));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += (1.0f - weight.x) * weight.y * (1.0f - weight.z) * v0.sdf;
                color_float = color_float + (1.0f - weight.x) * weight.y * (1.0f - weight.z) * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += (1.0f - weight.x) * weight.y * (1.0f - weight.z) * v.sdf;
                color_float = color_float + (1.0f - weight.x) * weight.y * (1.0f - weight.z) * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(0.0f, 0.0f, voxel_size));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += (1.0f - weight.x) * (1.0f - weight.y) * weight.z * v0.sdf;
                color_float = color_float + (1.0f - weight.x) * (1.0f - weight.y) * weight.z * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += (1.0f - weight.x) * (1.0f - weight.y) * weight.z * v.sdf;
                color_float = color_float + (1.0f - weight.x) * (1.0f - weight.y) * weight.z * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(voxel_size, voxel_size, 0.0f));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += weight.x * weight.y * (1.0f - weight.z) * v0.sdf;
                color_float = color_float + weight.x * weight.y * (1.0f - weight.z) * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += weight.x * weight.y * (1.0f - weight.z) * v.sdf;
                color_float = color_float + weight.x * weight.y * (1.0f - weight.z) * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(0.0f, voxel_size, voxel_size));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += (1.0f - weight.x) * weight.y * weight.z * v0.sdf;
                color_float = color_float + (1.0f - weight.x) * weight.y * weight.z * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += (1.0f - weight.x) * weight.y * weight.z * v.sdf;
                color_float = color_float + (1.0f - weight.x) * weight.y * weight.z * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(voxel_size, 0.0f, voxel_size));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += weight.x * (1.0f - weight.y) * weight.z * v0.sdf;
                color_float = color_float + weight.x * (1.0f - weight.y) * weight.z * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += weight.x * (1.0f - weight.y) * weight.z * v.sdf;
                color_float = color_float + weight.x * (1.0f - weight.y) * weight.z * vColor;
            }

            v = GetVoxel(pos_dual + make_float3(voxel_size, voxel_size, voxel_size));
            if (v.weight == 0) 
            {
                vColor = make_float3(v0.color.x, v0.color.y, v0.color.z);
                distance += weight.x * weight.y * weight.z * v0.sdf;
                color_float = color_float + weight.x * weight.y * weight.z * vColor;
            } 
            else 
            {
                vColor = make_float3(v.color.x, v.color.y, v.color.z);
                distance += weight.x * weight.y * weight.z * v.sdf;
                color_float = color_float + weight.x * weight.y * weight.z * vColor;
            }

            uchar3 color = make_uchar3(color_float.x, color_float.y, color_float.z);
            v.weight = v0.weight;
            v.sdf = distance;
            v.color = color;
            return v;
        }

        __host__ __device__ bool TsdfVolume::SetVoxel(float3 position, const Voxel &voxel) 
        {
            int3 block_position = WorldToBlock(position);
            int3 local_voxel = WorldToLocalVoxel(position);
            HashEntry entry = HashTable::FindHashEntry(block_position);
            if (entry.pointer == kFreeEntry) 
            {
                return false;
            }
            HashTable::voxel_blocks_[entry.pointer].at(local_voxel) = voxel;
            return true;
        }

        __host__ __device__ bool TsdfVolume::UpdateVoxel(float3 position, const Voxel &voxel) 
        {
            int3 block_position = WorldToBlock(position);
            int3 local_voxel = WorldToLocalVoxel(position);
            HashEntry entry = HashTable::FindHashEntry(block_position);
            if (entry.pointer == kFreeEntry) 
            {
                return false;
            }
            HashTable::voxel_blocks_[entry.pointer]
                    .at(local_voxel)
                    .Combine(voxel, options_.max_sdf_weight);
            return true;
        }


        __host__ __device__ bool TsdfVolume::UpdateSemanticVoxel(float3 position,  const Voxel &voxel) 
        {
            int3 block_position = WorldToBlock(position);
            int3 local_voxel = WorldToLocalVoxel(position);
            HashEntry entry = HashTable::FindHashEntry(block_position);
            if (entry.pointer == kFreeEntry) 
            {
                return false;
            }
            HashTable::voxel_blocks_[entry.pointer]
                    .at(local_voxel)
                    .SemanticCombine(voxel, options_.max_sdf_weight,InstanceColorList_gpu);
            return true;
        }

        __global__ void AllocateFromDepthKernel(TsdfVolume *volume, float *depth, RgbdSensor sensor, float4x4 transform) 
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int size = sensor.rows * sensor.cols;

        //    DEBUG_PRINT("AllocateFromDepthKernel, index = %d\n", index)

            float truncation_distance = volume->GetOptions().truncation_distance;
            float block_size = volume->GetOptions().block_size * volume->GetOptions().voxel_size;

            float3 start_pt = make_float3(transform.m14, transform.m24, transform.m34);
            for (int i = index; i < size; i += stride) 
            {
                if (depth[i] < volume->GetOptions().min_sensor_depth || depth[i] > volume->GetOptions().max_sensor_depth)
                    continue;
                float3 point_unproject = GetPoint3d(i, depth[i], sensor);
                float3 point = transform * point_unproject;
                if (point.x == 0 && point.y == 0 && point.z == 0) 
                    continue;
                // compute start and end of the ray
                float3 ray_direction = normalize(point - start_pt);
                float surface_distance = distance(start_pt, point);
                float3 ray_start = start_pt;
                float3 ray_end = start_pt + ray_direction * (surface_distance + truncation_distance);
                // traverse the ray discretely using the block size and allocate it
                // adapted from https://github.com/francisengelmann/fast_voxel_traversal/blob/master/main.cpp
                int3 block_start = make_int3(floor(ray_start.x / block_size),
                                                                            floor(ray_start.y / block_size),
                                                                            floor(ray_start.z / block_size));

                int3 block_end = make_int3( floor(ray_end.x / block_size),
                                                                            floor(ray_end.y / block_size),
                                                                            floor(ray_end.z / block_size));

                int3 block_position = block_start;
                int3 step = make_int3(  sign(ray_direction.x),
                                                                sign(ray_direction.y),
                                                                sign(ray_direction.z));

                float3 delta_t;
                delta_t.x = (ray_direction.x != 0) ? fabs(block_size / ray_direction.x) : FLT_MAX;
                delta_t.y = (ray_direction.y != 0) ? fabs(block_size / ray_direction.y) : FLT_MAX;
                delta_t.z = (ray_direction.z != 0) ? fabs(block_size / ray_direction.z) : FLT_MAX;

                float3 boundary = make_float3(   (block_position.x + static_cast<float>(step.x)) * block_size,
                                                                                    (block_position.y + static_cast<float>(step.y)) * block_size,
                                                                                    (block_position.z + static_cast<float>(step.z)) * block_size);

                float3 max_t;
                max_t.x = (ray_direction.x != 0)
                          ? (boundary.x - ray_start.x) / ray_direction.x
                          : FLT_MAX;
                max_t.y = (ray_direction.y != 0)
                          ? (boundary.y - ray_start.y) / ray_direction.y
                          : FLT_MAX;
                max_t.z = (ray_direction.z != 0)
                          ? (boundary.z - ray_start.z) / ray_direction.z
                          : FLT_MAX;

                int3 diff = make_int3(0, 0, 0);
                bool neg_ray = false;

                if (block_position.x != block_end.x && ray_direction.x < 0) 
                {
                    diff.x--;
                    neg_ray = true;
                }
                if (block_position.y != block_end.y && ray_direction.y < 0) 
                {
                    diff.y--;
                    neg_ray = true;
                }
                if (block_position.z != block_end.z && ray_direction.z < 0) 
                {
                    diff.z--;
                    neg_ray = true;
                }
                volume->AllocateBlock(block_position);

                if (neg_ray) 
                {
                    block_position = block_position + diff;
                    volume->AllocateBlock(block_position);
                }

                while (block_position.x != block_end.x || block_position.y != block_end.y || block_position.z != block_end.z) 
                {
                    if (max_t.x < max_t.y) 
                    {
                        if (max_t.x < max_t.z) 
                        {
                            block_position.x += step.x;
                            max_t.x += delta_t.x;
                        } 
                        else 
                        {
                            block_position.z += step.z;
                            max_t.z += delta_t.z;
                        }
                    } 
                    else 
                    {
                        if (max_t.y < max_t.z) 
                        {
                            block_position.y += step.y;
                            max_t.y += delta_t.y;
                        } 
                        else 
                        {
                            block_position.z += step.z;
                            max_t.z += delta_t.z;
                        }
                    }
                    volume->AllocateBlock(block_position);

                //    z++;
                //    if (z > 10000){
                //        DEBUG_PRINT("Index = %d, i = %d, z = %d, pos = (%d, %d, %d), end = (%d, %d, %d), depth = %f, block_start = (%d, %d, %d), ray_start = (%f, %f, %f), block_end = (%d, %d, %d), ray_end = (%f, %f, %f)\n",index, i, z,block_position.x, block_position.y, block_position.z,block_end.x, block_end.y, block_end.z,depth[i],block_start.x,block_start.y,block_start.z,ray_start.x,ray_start.y,ray_start.z,block_end.x, block_end.y, block_end.z, ray_end.x, ray_end.y, ray_end.z)
                //        DEBUG_PRINT("point = (%f, %f, %f), point_unproj = (%f, %f, %f), start_point = (%f, %f, %f), transform=(%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f)\n",point.x,point.y,point.z,point_unproject.x,point_unproject.y,point_unproject.z,start_pt.x,start_pt.y,start_pt.z,transform.entries[0],transform.entries[1],transform.entries[2],transform.entries[3],transform.entries[4],transform.entries[5],transform.entries[6],transform.entries[7],transform.entries[8],transform.entries[9],transform.entries[10],transform.entries[11],transform.entries[12],transform.entries[13],transform.entries[14],transform.entries[15])
                //    }
                }
            }

//            DEBUG_PRINT("AllocateFromDepthKernel done, index = %d\n", index)
        }

        __global__ void IntegrateScanKernel(TsdfVolume *volume, uchar3 *color,
                                                                                      float *depth, RgbdSensor sensor,
                                                                                      float4x4 transform, float4x4 inv_transform) 
        {
            //loop through ALL entries
            //  if entry is in camera frustum
            //    loop through voxels inside block
            //    if voxel is in truncation region
            //      update voxels
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;

            int block_size = volume->GetOptions().block_size;
            float voxel_size = volume->GetOptions().voxel_size;
            float truncation_distance = volume->GetOptions().truncation_distance;

            for (int i = index; i < volume->GetNumEntries(); i += stride) 
            {
                float3 position = make_float3(  volume->GetHashEntry(i).position.x * voxel_size * block_size,
                                                                                volume->GetHashEntry(i).position.y * voxel_size * block_size,
                                                                                volume->GetHashEntry(i).position.z * voxel_size * block_size);
                // To camera coordinates
                float3 position_cam = inv_transform * position;
                // If behind camera plane discard
                if (position_cam.z < 0) 
                    continue;
                float3 block_center = make_float3(position_cam.x + 0.5 * voxel_size * block_size,
                                                                                        position_cam.y + 0.5 * voxel_size * block_size,
                                                                                        position_cam.z + 0.5 * voxel_size * block_size);
                int2 image_position = Project(block_center, sensor);
                if (image_position.x >= 0 && image_position.y >= 0 && image_position.x < sensor.cols && image_position.y < sensor.rows) 
                {
                    float3 start_pt = make_float3(0, 0, 0);

                    for (int bx = 0; bx < block_size; bx++) 
                    {
                        for (int by = 0; by < block_size; by++) 
                        {
                            for (int bz = 0; bz < block_size; bz++) 
                            {
                                float3 voxel_position = make_float3( position.x + bx * voxel_size,
                                                                                                            position.y + by * voxel_size,
                                                                                                            position.z + bz * voxel_size);
                                voxel_position = inv_transform * voxel_position;
                                image_position = Project(voxel_position, sensor);
                                // Check again inside the block
                                if (image_position.x >= 0 && image_position.y >= 0 && image_position.x < sensor.cols && image_position.y < sensor.rows) 
                                {
                                    int idx = image_position.y * sensor.cols + image_position.x;
                                    if (depth[idx] <= 0) 
                                        continue;
                                    if (depth[idx] < volume->GetOptions().min_sensor_depth) 
                                        continue;
                                    if (depth[idx] > volume->GetOptions().max_sensor_depth) 
                                        continue;
                                    float3 point3d = GetPoint3d(idx, depth[idx], sensor);
                                    float surface_distance = distance(start_pt, point3d);
                                    float voxel_distance = distance(start_pt, voxel_position);
                                    if (voxel_distance > surface_distance - truncation_distance && voxel_distance < surface_distance + truncation_distance && (depth[idx] < volume->GetOptions().max_sensor_depth)) 
                                    {
                                        Voxel voxel;
                                        voxel.sdf = surface_distance - voxel_distance;
                                        voxel.color = color[idx];
                                        voxel.weight = (unsigned char) 1;
                                        // To world coordinates
                                        voxel_position = transform * voxel_position;
                                        volume->UpdateVoxel(voxel_position, voxel);
                                    } 
                                    else if (voxel_distance < surface_distance - truncation_distance) 
                                    {
                                        voxel_position = transform * voxel_position;
                                        Voxel voxel;
                                        voxel.sdf = truncation_distance;
                                        voxel.color = color[idx];
                                        voxel.weight = (unsigned char) 1;
                                        volume->UpdateVoxel(voxel_position, voxel);
                                    }
                                }
                            }
                        }
                    }  // End single block update
                }
            }
        }

        void TsdfVolume::IntegrateScanAsync(
                const RgbdSensor &sensor,
                unsigned char *bgr,
                float *depth,
                float4x4 const &camera_pose) 
        {
            if (should_call_next_ != "IntegrateScanAsync") 
            {
                std::cerr << "Please call the functions like Integration -> RenderAsync -> GetRenderResults."
                          << " You should have called " << should_call_next_ << std::endl;
                exit(EXIT_FAILURE);
            }
            should_call_next_ = "RenderAsync";

            float4x4 inv_camera_pose = camera_pose.getInverse();
            int threads_per_block = THREADS_PER_BLOCK2;
            int thread_blocks = (options_.num_buckets * options_.bucket_size + threads_per_block - 1) / threads_per_block;

            if (int_event_) 
            {
                gpuErrchk(hipEventSynchronize(int_event_->cpy_dth))
                eventDestroy(int_event_);
            } 
            else 
            {
                int_event_ = new Event;
            }
            eventCreate(int_event_);

            // Copy inputs to page-locked memory
            memcpy((void *) h_bgr_in_, (void *) bgr, sizeof(uchar3) * num_pixels());
            memcpy(h_depth_in_, depth, sizeof(float) * num_pixels());

            // Copy mem to device
            gpuErrchk(hipMemcpyAsync(d_bgr_in_, h_bgr_in_, sizeof(uchar3) * num_pixels(), hipMemcpyHostToDevice, int_stream_))
            gpuErrchk(hipMemcpyAsync(d_depth_in_, h_depth_in_, sizeof(float) * num_pixels(), hipMemcpyHostToDevice, int_stream_))
            gpuErrchk(hipEventRecord(int_event_->cpy_htd, int_stream_))

            // Potentially wait for last rendering to complete
            for (auto& x : render_events_) 
            {
                if (x) 
                {
                    gpuErrchk(hipStreamWaitEvent(int_stream_, x->cpy_dth, 0));
                }
            }

            // Call Kernels
            int mem = 0;
            DEBUG_PRINT("Launching AllocateFromDepthKernel kernel\n")
            clock_t started = clock();

            if (camera_pose.entries2[0][0] > 10.0f || camera_pose.entries2[0][0] < -10.0f) 
            {
                std::cout << "TRANSFORM!!! " << std::endl;
                for (int r = 0; r < 4; r++) 
                {
                    for (int c = 0; c < 4; c++)
                        std::cout << camera_pose.entries2[r][c] << " ";
                    std::cout << std::endl;
                }
            }

            AllocateFromDepthKernel<<<thread_blocks, threads_per_block, mem, int_stream_>>>(this, d_depth_in_, sensor, camera_pose);
            #ifdef DR_FUSION_DEBUG_SYNC_LAUNCH
                // TODO: This one throws: an illegal memory access was encountered
                gpuErrchk( hipPeekAtLastError() )

                // TODO(lukas)
                // gpuErrchk( hipDeviceSynchronize() )
                gpuErrchk( hipStreamSynchronize(int_stream_) )
                clock_t ended = clock();
                double MilliSecondsTaken = 1000.0f*(ended-started)/(float)(CLOCKS_PER_SEC);
                DEBUG_PRINT("Sync After AllocateFromDepthKernel kernel: %f ms\n", MilliSecondsTaken)
            #endif

            IntegrateScanKernel<<<thread_blocks, threads_per_block, mem, int_stream_>>>(this, d_bgr_in_, d_depth_in_, sensor, camera_pose,inv_camera_pose);
            #ifdef DR_FUSION_DEBUG_SYNC_LAUNCH
                gpuErrchk( hipPeekAtLastError() )
                // TODO(lukas)
                // gpuErrchk( hipDeviceSynchronize() )
                gpuErrchk( hipStreamSynchronize(int_stream_) )
            #endif
            gpuErrchk(hipEventRecord(int_event_->compute, int_stream_))
            gpuErrchk(hipEventRecord(int_event_->cpy_dth, int_stream_))
        }



        __global__ void IntegrateScanWithSemanticKernel(TsdfVolume *volume, uchar3 *color,
                                            float *depth, RgbdSensor sensor,unsigned char* masks, int masks_num,
                                            float4x4 transform, float4x4 inv_transform) 
        {
            //loop through ALL entries
            //  if entry is in camera frustum
            //    loop through voxels inside block
            //    if voxel is in truncation region
            //      update voxels
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;

            int block_size = volume->GetOptions().block_size;
            float voxel_size = volume->GetOptions().voxel_size;
            float truncation_distance = volume->GetOptions().truncation_distance;

            for (int i = index; i < volume->GetNumEntries(); i += stride) 
            {
                float3 position = make_float3(  volume->GetHashEntry(i).position.x * voxel_size * block_size,
                                                                                volume->GetHashEntry(i).position.y * voxel_size * block_size,
                                                                                volume->GetHashEntry(i).position.z * voxel_size * block_size);
                // To camera coordinates
                float3 position_cam = inv_transform * position;
                // If behind camera plane discard
                if (position_cam.z < 0) 
                    continue;
                float3 block_center = make_float3(position_cam.x + 0.5 * voxel_size * block_size,
                                                                                        position_cam.y + 0.5 * voxel_size * block_size,
                                                                                        position_cam.z + 0.5 * voxel_size * block_size);
                int2 image_position = Project(block_center, sensor);

                if (image_position.x >= 0 && image_position.y >= 0 &&image_position.x < sensor.cols && image_position.y < sensor.rows) 
                {
                    float3 start_pt = make_float3(0, 0, 0);
                    for (int bx = 0; bx < block_size; bx++) 
                    {
                        for (int by = 0; by < block_size; by++) 
                        {
                            for (int bz = 0; bz < block_size; bz++) 
                            {
                                float3 voxel_position = make_float3( position.x + bx * voxel_size,
                                                                                                            position.y + by * voxel_size,
                                                                                                            position.z + bz * voxel_size);
                                voxel_position = inv_transform * voxel_position;
                                image_position = Project(voxel_position, sensor);
                                // Check again inside the block
                                if (image_position.x >= 0 && image_position.y >= 0 &&image_position.x < sensor.cols &&image_position.y < sensor.rows) 
                                {
                                    int idx = image_position.y * sensor.cols + image_position.x;
                                    if (depth[idx] <= 0) 
                                        continue;
                                    if (depth[idx] < volume->GetOptions().min_sensor_depth) 
                                        continue;
                                    if (depth[idx] > volume->GetOptions().max_sensor_depth) 
                                        continue;
                                    float3 point3d = GetPoint3d(idx, depth[idx], sensor);
                                    float surface_distance = distance(start_pt, point3d);
                                    float voxel_distance = distance(start_pt, voxel_position);
                                    
                                    Voxel voxel;
                                    for(int mask_idx = 0;mask_idx<masks_num;mask_idx++)
                                    {
                                        unsigned char maskPixelValue =  masks[sensor.cols * sensor.rows * mask_idx + idx];
                                        if (maskPixelValue != 0)
                                        {
                                            voxel.InstanceIndex = maskPixelValue;
                                        }
                                    }


                                    if (voxel_distance > surface_distance - truncation_distance &&voxel_distance < surface_distance + truncation_distance &&(depth[idx] < volume->GetOptions().max_sensor_depth)) 
                                    {
                                        voxel.sdf = surface_distance - voxel_distance;
                                        voxel.color = color[idx];
                                        voxel.weight = (unsigned char) 1;
                                        // To world coordinates
                                        voxel_position = transform * voxel_position;
                                        volume->UpdateSemanticVoxel(voxel_position, voxel);
                                    } 
                                    else if (voxel_distance < surface_distance - truncation_distance) 
                                    {
                                        voxel_position = transform * voxel_position;
                                        voxel.sdf = truncation_distance;
                                        voxel.color = color[idx];
                                        voxel.weight = (unsigned char) 1;
                                        volume->UpdateSemanticVoxel(voxel_position, voxel);
                                    }
                                }
                            }
                        }
                    }  // End single block update
                }
            }
        }




        void TsdfVolume::IntegrateScanWithSemanticAsync(
                        const RgbdSensor &sensor,
                        unsigned char *bgr,
                        float *depth,
                        unsigned char* masks, 
                        int masks_num,
                        float4x4 const &camera_pose) 
                {
                    if (should_call_next_ != "IntegrateScanAsync") 
                    {
                        std::cerr << "Please call the functions like Integration -> RenderAsync -> GetRenderResults."
                                << " You should have called " << should_call_next_ << std::endl;
                        exit(EXIT_FAILURE);
                    }
                    should_call_next_ = "RenderAsync";

                    float4x4 inv_camera_pose = camera_pose.getInverse();
                    int threads_per_block = THREADS_PER_BLOCK2;
                    int thread_blocks =(options_.num_buckets * options_.bucket_size + threads_per_block - 1) /threads_per_block;

                    if (int_event_) 
                    {
                        gpuErrchk(hipEventSynchronize(int_event_->cpy_dth))
                        eventDestroy(int_event_);
                    } 
                    else 
                    {
                        int_event_ = new Event;
                    }
                    eventCreate(int_event_);

                    // Copy inputs to page-locked memory
                    memcpy((void *) h_bgr_in_, (void *) bgr, sizeof(uchar3) * num_pixels());
                    memcpy(h_depth_in_, depth, sizeof(float) * num_pixels());

                    // Copy mem to device
                    gpuErrchk(hipMemcpyAsync(d_bgr_in_, h_bgr_in_, sizeof(uchar3) * num_pixels(),hipMemcpyHostToDevice, int_stream_))
                    gpuErrchk(hipMemcpyAsync(d_depth_in_, h_depth_in_, sizeof(float) * num_pixels(),hipMemcpyHostToDevice, int_stream_))
                    gpuErrchk(hipEventRecord(int_event_->cpy_htd, int_stream_))

                    // Potentially wait for last rendering to complete
                    for (auto& x : render_events_) 
                    {
                        if (x) 
                        {
                            gpuErrchk(hipStreamWaitEvent(int_stream_, x->cpy_dth, 0));
                        }
                    }

                    // Call Kernels
                    int mem = 0;
                    DEBUG_PRINT("Launching AllocateFromDepthKernel kernel\n")
                    clock_t started = clock();

                    if (camera_pose.entries2[0][0] > 10.0f || camera_pose.entries2[0][0] < -10.0f) 
                    {
                        std::cout << "TRANSFORM!!! " << std::endl;
                        for (int r = 0; r < 4; r++) 
                        {
                            for (int c = 0; c < 4; c++)
                                std::cout << camera_pose.entries2[r][c] << " ";
                            std::cout << std::endl;
                        }
                    }


                    AllocateFromDepthKernel<<<thread_blocks, threads_per_block, mem, int_stream_>>>(this, d_depth_in_, sensor, camera_pose);
                    #ifdef DR_FUSION_DEBUG_SYNC_LAUNC
                        // TODO: This one throws: an illegal memory access was encountered
                        gpuErrchk( hipPeekAtLastError() )
                        // TODO(lukas)
                        // gpuErrchk( hipDeviceSynchronize() )
                        gpuErrchk( hipStreamSynchronize(int_stream_) )
                        clock_t ended = clock();
                        double MilliSecondsTaken = 1000.0f*(ended-started)/(float)(CLOCKS_PER_SEC);
                        DEBUG_PRINT("Sync After AllocateFromDepthKernel kernel: %f ms\n", MilliSecondsTaken)
                    #endif
                    IntegrateScanWithSemanticKernel<<<thread_blocks, threads_per_block, mem, int_stream_>>>(
                            this, d_bgr_in_, d_depth_in_, sensor, masks, masks_num,camera_pose,
                            inv_camera_pose);
                    // IntegrateScanKernel<<<thread_blocks, threads_per_block, mem, int_stream_>>>(
                    //         this, d_bgr_in_, d_depth_in_, sensor, camera_pose,
                    //         inv_camera_pose);

                    #ifdef DR_FUSION_DEBUG_SYNC_LAUNCH
                        gpuErrchk( hipPeekAtLastError() )
                        // TODO(lukas)
                        // gpuErrchk( hipDeviceSynchronize() )
                        // std::cout<<"IntegrateScanWithSemanticKernel sync"<<std::endl;
                        gpuErrchk( hipStreamSynchronize(int_stream_) )
                    #endif
                    gpuErrchk(hipEventRecord(int_event_->compute, int_stream_))
                    gpuErrchk(hipEventRecord(int_event_->cpy_dth, int_stream_))
                }


        __global__ void GenerateRgbDepthKernel(TsdfVolume *volume, 
                                                                                                RgbdSensor sensor,
                                                                                                float4x4 camera_pose, 
                                                                                                uchar3 *virtual_rgb, 
                                                                                                float *virtual_depth) 
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int size = sensor.rows * sensor.cols;

            float3 start_pt = make_float3(camera_pose.m14, camera_pose.m24, camera_pose.m34);
            for (int i = index; i < size; i += stride) 
            {
                float current_depth = 0;
                while (current_depth < volume->GetOptions().max_sensor_depth) 
                {
                    float3 point = GetPoint3d(i, current_depth, sensor);
                    point = camera_pose * point;
                    Voxel v = volume->GetInterpolatedVoxel(point);
                    if (v.weight == 0) 
                    {
                        current_depth += volume->GetOptions().truncation_distance;
                    } 
                    else 
                    {
                        current_depth += v.sdf;
                    }
                    if (v.weight != 0 && v.sdf < volume->GetOptions().voxel_size) 
                        break;
                }
                if (current_depth < volume->GetOptions().max_sensor_depth) 
                {
                    float3 point = GetPoint3d(i, current_depth, sensor);
                    point = camera_pose * point;
                    Voxel v = volume->GetInterpolatedVoxel(point);
                    virtual_rgb[i] = v.color;
                    virtual_depth[i] = current_depth;
                } 
                else 
                {
                    virtual_rgb[i] = make_uchar3(0, 0, 0);
                    virtual_depth[i] = 0.0;
                }
            }
        }

        void TsdfVolume::RenderAsync(std::vector<float4x4> camera_poses, RgbdSensor sensor) 
        {
            if (should_call_next_ != "RenderAsync") {
                std::cerr << "Please call the functions like IntegrateScanAsync -> RenderAsync -> GetRenderResult."
                          << " You should have called " << should_call_next_ << std::endl;
                exit(EXIT_FAILURE);
            }
            should_call_next_ = "GetRenderResult";

            // Check input
            if (render_streams_.size() != camera_poses.size()) 
            {
                std::cerr << "Can only render exactly as many poses as streams."
                          << " Streams: " << std::to_string(render_streams_.size())
                          << ", Poses: " << std::to_string(camera_poses.size()) << "." << std::endl;
                exit(EXIT_FAILURE);
            }

            if ((sensor.rows != options_.height) || (sensor.cols != options_.width)) 
            {
                std::cerr << "Image sizes don't match." << std::endl;
                exit(EXIT_FAILURE);
            }

            // Wait for all rendering to complete (because we reuse device buffers)
            for (auto& x: render_events_)
            {
                if (x)
                {
                    hipEventSynchronize(render_events_[0]->cpy_dth);
                    eventDestroy(x);
                }
                else
                {
                    x = new Event;
                }

                eventCreate(x);
            }

            // Kernel options
            int threads_per_block = THREADS_PER_BLOCK2;
            int thread_blocks = DIV_UP(num_pixels(), threads_per_block);
            int mem = 0;

            // Start Kernel
            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                GenerateRgbDepthKernel<<<thread_blocks, threads_per_block, mem, render_streams_[i]>>>(this, sensor, camera_poses[i], d_bgr_render_[i], d_depth_render_[i]);
                hipEventRecord(render_events_[i]->cpy_htd, render_streams_[i]);
                hipEventRecord(render_events_[i]->compute, render_streams_[i]);
            }

            // Copy memory back
            auto &h_bgr_free = h_bgr_render_free();
            auto &h_depth_free = h_depth_render_free();
            // TODO: a sync here seems to solve the problem (no it does not)
            gpuErrchk( hipDeviceSynchronize() );

            for (int i = 0; i < options_.num_render_streams; i++) 
            {
              gpuErrchk(hipMemcpyAsync(h_bgr_free[i], d_bgr_render_[i], sizeof(uchar3) * sensor.rows * sensor.cols, hipMemcpyDeviceToHost, render_streams_[i]));
              gpuErrchk( hipDeviceSynchronize() ); // This throws
              gpuErrchk(hipMemcpyAsync(h_depth_free[i], d_depth_render_[i], sizeof(float) * sensor.rows * sensor.cols, hipMemcpyDeviceToHost, render_streams_[i]));
              gpuErrchk( hipDeviceSynchronize() );
              gpuErrchk(hipEventRecord(render_events_[i]->cpy_dth, render_streams_[i]) );
              gpuErrchk( hipDeviceSynchronize() );
            }

            // TODO: remove for perf.
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() ); // This throws
        }



        __global__ void GenerateRgbDepthMaskKernel(TsdfVolume *volume, 
                                                                                                            RgbdSensor sensor, 
                                                                                                            float4x4 camera_pose, 
                                                                                                            uchar3 *virtual_rgb, 
                                                                                                            uchar3 *virtual_instance_rgb, 
                                                                                                            float *virtual_depth,
                                                                                                            unsigned char* InstanceMasks_gpu,
                                                                                                            int InstanceNum,
                                                                                                            unsigned char* InstanceColorList_gpu) 
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int stride = blockDim.x * gridDim.x;
            int size = sensor.rows * sensor.cols;

            float3 start_pt =make_float3(camera_pose.m14, camera_pose.m24, camera_pose.m34);
            for (int i = index; i < size; i += stride) 
            {
                float current_depth = 0;
                while (current_depth < volume->GetOptions().max_sensor_depth) 
                {
                    float3 point = GetPoint3d(i, current_depth, sensor);
                    point = camera_pose * point;
                    Voxel v = volume->GetInterpolatedVoxel(point);
                    if (v.weight == 0) 
                    {
                        current_depth += volume->GetOptions().truncation_distance;
                    } 
                    else 
                    {
                        current_depth += v.sdf;
                    }
                    if (v.weight != 0 && v.sdf < volume->GetOptions().voxel_size) 
                        break;
                }
                if (current_depth < volume->GetOptions().max_sensor_depth) 
                {
                    float3 point = GetPoint3d(i, current_depth, sensor);
                    point = camera_pose * point;
                    Voxel v = volume->GetInterpolatedVoxel(point);
                    virtual_rgb[i] = v.color;
                    if(v.InstanceIndex !=0 && v.InstanceFrameNum>=0)
                    {
                        virtual_instance_rgb[i] = make_uchar3(InstanceColorList_gpu[3*(v.InstanceIndex-1)+0],
                                                                                                          InstanceColorList_gpu[3*(v.InstanceIndex-1)+1],
                                                                                                          InstanceColorList_gpu[3*(v.InstanceIndex-1)+2]);
                        // InstanceMasks_gpu[(v.InstanceIndex-1)*size+i] = 1;
                        InstanceMasks_gpu[(v.InstanceIndex-1)*size+i] = 1;
                    }
                    else
                    {
                        virtual_instance_rgb[i] = make_uchar3(200,200,200);
                    }

                    // if(v.InstanceIndex!=0)
                    // {
                        // InstanceMasks_gpu[(v.InstanceIndex-1)*size+i] = 255;
                    // }
                    // else
                    // {
                    //     InstanceMasks_gpu[(v.InstanceIndex-1)*size+i] = 0; 
                    // }
                    virtual_depth[i] = current_depth;
                }
                else 
                {
                    virtual_rgb[i] = make_uchar3(0, 0, 0);
                    virtual_instance_rgb[i] = make_uchar3(0, 0, 0);
                    virtual_depth[i] = 0.0;
                }
            }
        }


        void TsdfVolume::RenderWithMaskAsync(std::vector<float4x4> camera_poses, RgbdSensor sensor) 
        {
            if (should_call_next_ != "RenderAsync") 
            {
                std::cerr << "Please call the functions like IntegrateScanAsync -> RenderAsync -> GetRenderResult."
                          << " You should have called " << should_call_next_ << std::endl;
                exit(EXIT_FAILURE);
            }
            should_call_next_ = "GetRenderResult";
            // Check input
            if (render_streams_.size() != camera_poses.size()) 
            {
                std::cerr << "Can only render exactly as many poses as streams."
                          << " Streams: " << std::to_string(render_streams_.size())
                          << ", Poses: " << std::to_string(camera_poses.size()) << "." << std::endl;
                exit(EXIT_FAILURE);
            }

            if ((sensor.rows != options_.height) || (sensor.cols != options_.width)) 
            {
                std::cerr << "Image sizes don't match." << std::endl;
                exit(EXIT_FAILURE);
            }

            // Wait for all rendering to complete (because we reuse device buffers)
            for (auto& x: render_events_)
            {
                if (x)
                {
                    hipEventSynchronize(render_events_[0]->cpy_dth);
                    eventDestroy(x);
                }
                else
                {
                    x = new Event;
                }
                eventCreate(x);
            }

            // Kernel options
            int threads_per_block = THREADS_PER_BLOCK2;
            int thread_blocks = DIV_UP(num_pixels(), threads_per_block);
            int mem = 0;

            // Start Kernel
            // std::cout<<"options_.num_render_streams:"<<options_.num_render_streams<<std::endl;
            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                // gpuErrchk(hipMalloc(&InstanceMasks_gpu,options_.height*options_.width*GetInstanceNum() *sizeof(unsigned char)));
                gpuErrchk(hipHostMalloc(&InstanceMasks_gpu,options_.height*options_.width*GetInstanceNum() *sizeof(unsigned char)));
                // hipHostMalloc(void **ptr, size_t size);
                gpuErrchk(hipMemset(InstanceMasks_gpu, 0, options_.height*options_.width*GetInstanceNum() *sizeof(unsigned char)));

                GenerateRgbDepthMaskKernel<<<thread_blocks, threads_per_block, mem, render_streams_[i]>>>(this, sensor, camera_poses[i], d_bgr_render_[i], d_instance_bgr_render_[i], d_depth_render_[i],InstanceMasks_gpu,GetInstanceNum(),InstanceColorList_gpu);

                hipEventRecord(render_events_[i]->cpy_htd, render_streams_[i]);
                hipEventRecord(render_events_[i]->compute, render_streams_[i]);
            }

            // Copy memory back
            auto &h_bgr_free = h_bgr_render_free();
            auto &h_instance_bgr_free = h_instance_bgr_render_free();
            auto &h_depth_free = h_depth_render_free();
            // TODO: a sync here seems to solve the problem (no it does not)
            gpuErrchk(hipDeviceSynchronize());

            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                gpuErrchk(hipMemcpyAsync(h_bgr_free[i], d_bgr_render_[i], sizeof(uchar3) * sensor.rows * sensor.cols,hipMemcpyDeviceToHost, render_streams_[i]));
                gpuErrchk(hipDeviceSynchronize()); // This throws

                gpuErrchk(hipMemcpyAsync(h_instance_bgr_free[i], d_instance_bgr_render_[i], sizeof(uchar3) * sensor.rows * sensor.cols,hipMemcpyDeviceToHost, render_streams_[i]));
                gpuErrchk(hipDeviceSynchronize()); // This throws

                gpuErrchk(hipMemcpyAsync(h_depth_free[i], d_depth_render_[i], sizeof(float) * sensor.rows * sensor.cols,hipMemcpyDeviceToHost, render_streams_[i]));
                gpuErrchk(hipDeviceSynchronize());

                gpuErrchk(hipEventRecord(render_events_[i]->cpy_dth, render_streams_[i]) );
                gpuErrchk(hipDeviceSynchronize());
            }

            

            // TODO: remove for perf.
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize()); // This throws
        }


        void TsdfVolume::GetRenderResult(std::vector<unsigned char *> &bgr, std::vector<float *> &depth) 
        {
            if (should_call_next_ != "GetRenderResult") 
            {
                std::cerr << "Please call the functions in a loop: IntegrateScanAsync -> RenderAsync -> GetRenderResult"
                          << ". You should have called " << should_call_next_ << std::endl;
                exit(EXIT_FAILURE);
            }
            should_call_next_ = "IntegrateScanAsync";

            if ((!bgr.empty()) || (!depth.empty())) 
            {
                std::cerr << "Input vectors must be empty." << std::endl;
                exit(EXIT_FAILURE);
            }

            // Wait for all renderings
            for (auto &x: render_events_)
                eventSynchronize(x);

            // Change blocked vs free state
            if (render_blocked_ == 0)
                render_blocked_ = 1;
            else
                render_blocked_ = 0;

            // Get results
            auto &h_bgr_blocked = h_bgr_render_blocked();
            auto &h_depth_blocked = h_depth_render_blocked();

            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                bgr.push_back((unsigned char *) h_bgr_blocked[i]);
                depth.push_back(h_depth_blocked[i]);
            }

            // TODO: remove for perf.
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        }

        void TsdfVolume::GetRenderWithMaskResult(std::vector<unsigned char *> &bgr, std::vector<unsigned char *> &instance_bgr,std::vector<float *> &depth,std::vector<cv::Mat>& InstanceMasks) 
        {
            if (should_call_next_ != "GetRenderResult") 
            {
                std::cerr << "Please call the functions in a loop: IntegrateScanAsync -> RenderAsync -> GetRenderResult"
                          << ". You should have called " << should_call_next_ << std::endl;
                exit(EXIT_FAILURE);
            }
            should_call_next_ = "IntegrateScanAsync";

            if ((!bgr.empty()) || (!depth.empty())) 
            {
                std::cerr << "Input vectors must be empty." << std::endl;
                exit(EXIT_FAILURE);
            }

            // Wait for all renderings
            for (auto &x: render_events_)
                eventSynchronize(x);

            // Change blocked vs free state
            if (render_blocked_ == 0)
                render_blocked_ = 1;
            else
                render_blocked_ = 0;

            // Get results
            auto &h_bgr_blocked = h_bgr_render_blocked();
            auto &h_instance_bgr_blocked = h_instance_bgr_render_blocked();
            auto &h_depth_blocked = h_depth_render_blocked();

            for (int i = 0; i < options_.num_render_streams; i++) 
            {
                bgr.push_back((unsigned char *) h_bgr_blocked[i]);
                instance_bgr.push_back((unsigned char *) h_instance_bgr_blocked[i]);
                depth.push_back(h_depth_blocked[i]);
            }
            for(int i = 0;i<GetInstanceNum();i++)
            {
                cv::Mat instanceMask(options_.height,options_.width,CV_8UC1);
                // hipMemcpy(instanceMask.data, &InstanceMasks_gpu[i*options_.height,options_.width],  options_.height*options_.width* sizeof(unsigned char), hipMemcpyHostToDevice);
                hipMemcpy(instanceMask.data, &InstanceMasks_gpu[i*options_.height*options_.width],  options_.height*options_.width* sizeof(unsigned char), hipMemcpyDeviceToHost);
                InstanceMasks.push_back(instanceMask);
            }
            
            

            // TODO: remove for perf.
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
            gpuErrchk(hipHostFree(InstanceMasks_gpu));
        }

        Mesh TsdfVolume::ExtractMesh(const float3 &lower_corner, const float3 &upper_corner) 
        {
            // TODO: Dirty Hack
            // Wait for all renderings
            for (auto &x: render_events_)
                eventSynchronize(x);

            MeshExtractor *mesh_extractor;
            hipMallocManaged(&mesh_extractor, sizeof(MeshExtractor));
            mesh_extractor->Init(20000000, options_.voxel_size);
            mesh_extractor->ExtractMesh(this, lower_corner, upper_corner);
            Mesh *mesh;
            hipMallocManaged(&mesh, sizeof(Mesh));
            *mesh = mesh_extractor->GetMesh();

            // TODO: Is this needed
            hipFree(mesh_extractor);
            return *mesh;
        }

        Mesh TsdfVolume::ExtractMeshWithInstance(const float3 &lower_corner, const float3 &upper_corner,bool is_instance) 
        {
            // TODO: Dirty Hack
            // Wait for all renderings
            for (auto &x: render_events_)
                eventSynchronize(x);

            MeshExtractor *mesh_extractor;
            hipMallocManaged(&mesh_extractor, sizeof(MeshExtractor));
            mesh_extractor->Init(20000000, options_.voxel_size);
            // mesh_extractor->ExtractMesh(this, lower_corner, upper_corner);
            mesh_extractor->ExtractMeshWithInstance(this, lower_corner, upper_corner,is_instance,InstanceColorList_gpu );
            Mesh *mesh;
            hipMallocManaged(&mesh, sizeof(Mesh));
            *mesh = mesh_extractor->GetMesh();

            // TODO: Is this needed
            hipFree(mesh_extractor);
            return *mesh;
        }


        void TsdfVolume::ExtractMeshAsync(const float3 &lower_corner, const float3 &upper_corner) 
        {
            if (should_call_next_ != "IntegrateScanAsync") 
            {
                std::cerr << "Please call this functions after GetRenderResult" << std::endl;
                exit(EXIT_FAILURE);
            }
           // TODO: Dirty Hack and should be unnecessary
           // Wait for all renderings
            // for (auto &x: render_events_)
            //     eventSynchronize(x);

            if (mesh_extractor != nullptr) 
            {
                std::cerr << "mesh_extractor should be NULL" << std::endl;
                exit(EXIT_FAILURE);
            }

            hipMallocManaged(&mesh_extractor, sizeof(MeshExtractor));

            mesh_extractor->Init(20000000, options_.voxel_size);
            mesh_extractor->ExtractMesh(this, lower_corner, upper_corner);
        }

        void TsdfVolume::ExtractMeshWithInstanceAsync(const float3 &lower_corner, const float3 &upper_corner,bool is_instance) 
        {
            if (should_call_next_ != "IntegrateScanAsync") 
            {
                std::cerr << "Please call this functions after GetRenderResult" << std::endl;
                exit(EXIT_FAILURE);
            }
           // TODO: Dirty Hack and should be unnecessary
           // Wait for all renderings
            // for (auto &x: render_events_)
            //     eventSynchronize(x);

            if (mesh_extractor != nullptr) 
            {
                std::cerr << "mesh_extractor should be NULL" << std::endl;
                exit(EXIT_FAILURE);
            }

            hipMallocManaged(&mesh_extractor, sizeof(MeshExtractor));

            mesh_extractor->Init(20000000, options_.voxel_size);
            mesh_extractor->ExtractMeshWithInstance(this, lower_corner, upper_corner,is_instance,InstanceColorList_gpu );
        }




        void TsdfVolume::GetMeshSync(size_t num_max, size_t *num, float *vert, float *cols) 
        {
            if (should_call_next_ != "IntegrateScanAsync") 
            {
                std::cerr << "Please call this functions after GetRenderResult" << std::endl;
                exit(EXIT_FAILURE);
            }

            if (mesh_extractor == nullptr)
            {
                std::cerr << "mesh_extractor should not be NULL (did you call ExtractMeshAsync before)?" << std::endl;
                exit(EXIT_FAILURE);
            }

            // Synchronizes
            Mesh mesh = mesh_extractor->GetMesh();

            // Copy Data
            if (num_max < mesh.num_triangles_)
            {
                std::cerr << "Did not provide enough storage for mesh." << std::endl;
                exit(EXIT_FAILURE);
            }
            *num = 3 * mesh.num_triangles_;  // 1 triangle = 3 vert

            for (size_t i_tri = 0; i_tri < mesh.num_triangles_; i_tri++)
            {
                size_t idx;
                // v0
                idx = 9 * i_tri + 3*0;
                vert[idx + 0] = mesh.triangles_[i_tri].v0.position.x;
                vert[idx + 1] = mesh.triangles_[i_tri].v0.position.y;
                vert[idx + 2] = mesh.triangles_[i_tri].v0.position.z;

                cols[idx + 0] = mesh.triangles_[i_tri].v0.color.z;
                cols[idx + 1] = mesh.triangles_[i_tri].v0.color.y;
                cols[idx + 2] = mesh.triangles_[i_tri].v0.color.x;

                // v1
                idx = 9 * i_tri + 3*1;
                vert[idx + 0] = mesh.triangles_[i_tri].v1.position.x;
                vert[idx + 1] = mesh.triangles_[i_tri].v1.position.y;
                vert[idx + 2] = mesh.triangles_[i_tri].v1.position.z;

                cols[idx + 0] = mesh.triangles_[i_tri].v1.color.z;
                cols[idx + 1] = mesh.triangles_[i_tri].v1.color.y;
                cols[idx + 2] = mesh.triangles_[i_tri].v1.color.x;

                // v2
                idx = 9 * i_tri + 3*2;
                vert[idx + 0] = mesh.triangles_[i_tri].v2.position.x;
                vert[idx + 1] = mesh.triangles_[i_tri].v2.position.y;
                vert[idx + 2] = mesh.triangles_[i_tri].v2.position.z;

                cols[idx + 0] = mesh.triangles_[i_tri].v2.color.z;
                cols[idx + 1] = mesh.triangles_[i_tri].v2.color.y;
                cols[idx + 2] = mesh.triangles_[i_tri].v2.color.x;
            }

            // Free Memory
            mesh_extractor->Free();
            hipFree(mesh_extractor);
            mesh_extractor = nullptr;
        }

        __host__ __device__ TsdfVolumeOptions TsdfVolume::GetOptions() 
        {
            return options_;
        }


        __host__ std::vector<uchar3 *> &TsdfVolume::h_bgr_render_blocked() 
        {
            if (render_blocked_ == 0)
                return std::get<0>(h_bgr_render_);
            else
                return std::get<1>(h_bgr_render_);
        }

        __host__ std::vector<uchar3 *> &TsdfVolume::h_instance_bgr_render_blocked() 
        {
            if (render_blocked_ == 0)
                return std::get<0>(h_instance_bgr_render_);
            else
                return std::get<1>(h_instance_bgr_render_);
        }


        __host__ std::vector<float *> &TsdfVolume::h_depth_render_blocked() 
        {
            if (render_blocked_ == 0)
                return std::get<0>(h_depth_render_);
            else
                return std::get<1>(h_depth_render_);
        }

        __host__ std::vector<uchar3 *> &TsdfVolume::h_bgr_render_free() 
        {
            if (render_blocked_ == 0)
                return std::get<1>(h_bgr_render_);
            else
                return std::get<0>(h_bgr_render_);
        }

        __host__ std::vector<uchar3 *> &TsdfVolume::h_instance_bgr_render_free() 
        {
            if (render_blocked_ == 0)
                return std::get<1>(h_instance_bgr_render_);
            else
                return std::get<0>(h_instance_bgr_render_);
        }

        __host__ std::vector<float *> &TsdfVolume::h_depth_render_free() 
        {
            if (render_blocked_ == 0)
                return std::get<1>(h_depth_render_);
            else
                return std::get<0>(h_depth_render_);
        }

    }  // namespace tsdfvh

}  // namespace refusion
